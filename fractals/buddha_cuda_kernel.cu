#include "hip/hip_runtime.h"
#include "buddha_cuda_kernel.h"
#include <chrono>
#include <cmath>
#include <complex>
#include <iostream>
#include <random>
#include <string>
#include <utility>

#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <hip/hip_complex.h>

//emacs M-X c++-mode

//With cuda we need to check return codes often
#define PRINT_ON_SUCCESS (0)

void checkError(hipError_t code, char const * func, const char *file, const int line, bool abort)
{
    if (code != hipSuccess) 
    {
        const char * errorMessage = hipGetErrorString(code);
        fprintf(stderr, "CUDA error returned from \"%s\" at %s:%d, Error code: %d (%s)\n", func, file, line, code, errorMessage);
        if (abort){
            hipDeviceReset();
            exit(code);
        }
    }
    else if (PRINT_ON_SUCCESS)
    {
        const char * errorMessage = hipGetErrorString(code);
        fprintf(stderr, "CUDA error returned from \"%s\" at %s:%d, Error code: %d (%s)\n", func, file, line, code, errorMessage);
    }
}
 
void checkLastError(char const * func, const char *file, const int line, bool abort)
{
    hipError_t code = hipGetLastError();
    if (code != hipSuccess)
    {
        const char * errorMessage = hipGetErrorString(code);
        fprintf(stderr, "CUDA error returned from \"%s\" at %s:%d, Error code: %d (%s)\n", func, file, line, code, errorMessage);
        if (abort) {
            hipDeviceReset();
            exit(code);
        }
    }
    else if (PRINT_ON_SUCCESS)
    {
        const char * errorMessage = hipGetErrorString(code);
        fprintf(stderr, "CUDA error returned from \"%s\" at %s:%d, Error code: %d (%s)\n", func, file, line, code, errorMessage);
    }
}



// To be used around calls that return an error code, ex. hipDeviceSynchronize or hipMallocManaged
void checkError(hipError_t code, char const * func, const char *file, const int line, bool abort = true);
#define checkCUDAError(val) { checkError((val), #val, __FILE__, __LINE__); }    // in-line regular function
#define checkCUDAError2(val) check((val), #val, __FILE__, __LINE__) // typical macro 
 
// To be used after calls that do not return an error code, ex. kernels to check kernel launch errors
void checkLastError(char const * func, const char *file, const int line, bool abort = true);
#define checkLastCUDAError(func) { checkLastError(func, __FILE__, __LINE__); }
#define checkLastCUDAError_noAbort(func) { checkLastError(func, __FILE__, __LINE__, 0); }
 
using namespace std;

void cudaPrintDeviceProperties(hipDeviceProp_t & devProp) {
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
}

int cuda_info() {
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);

  if (nDevices == 0) {
    cout << "No CUDA Found" << endl;
  }
  else
  {
    for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      cudaPrintDeviceProperties(prop);
    }
  }

  return nDevices;
}


__global__ void vec_add_kernel(float *a, float *b, float *c, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

//Simple Test to see things are working
int cuda_vec_add(unsigned int w, unsigned int h) {
  cout << "CUDA Test: vector add" << endl;
  const int n = w*h;

  vector<float> h_a(n, 1.1);
  vector<float> h_b(n, 2.2);
  vector<float> h_c(n, 0.0);

  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, n*sizeof(float));
  hipMalloc(&d_b, n*sizeof(float));
  hipMalloc(&d_c, n*sizeof(float));

  hipMemcpy(d_a, &h_a[0], n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &h_b[0], n*sizeof(float), hipMemcpyHostToDevice);

  vec_add_kernel<<<((n-1)*256)/256 + 1,256>> >(d_a, d_b, d_c, n);

  hipMemcpy(&h_c[0], d_c, n*sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  cout << "First and last elements of vector should be 3.3: " << h_c[0] << " " << h_c[w*h - 1] << endl;
  
  return 0;
}


#if 0
__device__ bool skipInSettest(complex<double> sample) {
    if ((abs(sample - complex<double>(-1, 0)) < 0.25) ||
        (abs(1.0 - sqrt(1.0 - 4.0 * sample))) < 1.0)
      return true;
    return false;
  }
#endif


//prototype of what I will actually need
//2D array on device is one block of memory
__global__ void generate_hits_prototype_kernel(unsigned long long *rH, int w, int h) {
  int i = threadIdx.x + blockDim.x * blockIdx.x; //dim is 1 and tix is 1
  
  if (i >= h*w)
    return;

  //hiprandState state;
  hiprandStateMRG32k3a state;
  hiprand_init((unsigned long long)clock() + i, 0, 0, &state);

  //generate enough samples to overcome cuda overhead
  for (int sample_ix = 0; sample_ix < 1000; ++sample_ix)
  {
    double pr = hiprand_uniform_double (&state);
    double pi = hiprand_uniform_double (&state);

    //scale the random numbers
    pr = -2 + 3.0 *pr;
    pi = -1 + 2.0 *pi;

    hipDoubleComplex p = make_hipDoubleComplex(pr, pi);
    hipDoubleComplex c = hipCmul(p, p);

    //STL doesnt compile on devices
    //complex<double> sample(0.0,0.0);
    //if (true == skipInSettest(sample))
    //  printf("complex works");

    //do some rejection testing sort of like what we need for fractals
    
    c = hipCadd(hipCmul(c, c), p);

    if (hipCabs(c) < 0.5)
    {
      //See which box to put the hit in
      double minx = -2.0;
      double maxx = 1.0;
      double miny = -1.0;
      double maxy = 1.0;
      if ((hipCreal(c) <= maxx) && (hipCreal(c) >= minx) && (hipCimag(c) <= maxy) &&
          (hipCimag(c) >= miny)) {
        int x = ((hipCreal(c) - minx) * h) /
            (maxx - minx);
        int y = ((hipCimag(c) - miny) * w) /
            (maxy - miny);
	int ii = x + w*y;

	//printf("%d %f %f\n",i,hipCreal(c),hipCimag(c));
	if (ii < h*w)
	  atomicAdd(&rH[x + w*y], 1); //need atomics here -> in practice collisions should be very rare
      }
    }
    else
      continue;

  }
}

int cuda_generate_hits_prototype(unsigned int w, unsigned int h)
{
  cout << "CUDA Test: generate buddhabrot hits prototype" << endl;
  auto start = chrono::high_resolution_clock::now();
  vector<unsigned long long> redHits;

  redHits.resize(w*h,0);



  //Try several modes of parallelization
  //Mode 1
  unsigned long long * drH;

  checkCUDAError(hipMalloc(&drH, w*h*sizeof(redHits[0])));
  
  //copy from host to cuda memory
  
  checkCUDAError(hipMemcpy(drH, &redHits[0], w*h*sizeof(redHits[0]), hipMemcpyHostToDevice));
  
  generate_hits_prototype_kernel<<<256,256>>>(drH, w, h);
  
  checkLastCUDAError_noAbort("kernelA");		   
  
  checkCUDAError(hipMemcpy(&redHits[0], drH, w*h*sizeof(redHits[0]), hipMemcpyDeviceToHost));
     
  checkCUDAError(hipFree(drH));
  
  auto end = chrono::high_resolution_clock::now();
  //cout << "sample time " << tix << " " << chrono::duration_cast<chrono::milliseconds>(end - start).count() << " ms" << endl; 

   unsigned long long  hitsum = 0;
   for (auto e: redHits) {
       hitsum += e; }

   auto duration = chrono::duration_cast<chrono::milliseconds>(end - start).count();
   cout << "cuda generated pseudo hits using execution mode <<<256,256>>>: " << hitsum << " in " << duration <<
     " ms. hits per second: " <<  1000*hitsum/duration << endl;

   //Mode 2
   checkCUDAError(hipMalloc(&drH, w*h*sizeof(redHits[0])));
  
   //copy from host to cuda memory
   
   checkCUDAError(hipMemcpy(drH, &redHits[0], w*h*sizeof(redHits[0]), hipMemcpyHostToDevice));
  
   generate_hits_prototype_kernel<<<256*256,1>>>(drH, w, h);
   
   checkLastCUDAError_noAbort("kernelA");		   
   
   checkCUDAError(hipMemcpy(&redHits[0], drH, w*h*sizeof(redHits[0]), hipMemcpyDeviceToHost));
   
   checkCUDAError(hipFree(drH));
  
   end = chrono::high_resolution_clock::now();
   //cout << "sample time " << tix << " " << chrono::duration_cast<chrono::milliseconds>(end - start).count() << " ms" << endl; 
   
   hitsum = 0;
   for (auto e: redHits) {
     hitsum += e; }

   duration = chrono::duration_cast<chrono::milliseconds>(end - start).count();
   cout << "cuda generated pseudo hits using execution mode <<<256*256,1>>>: " << hitsum << " in " << duration <<
       " ms. hits per second: " <<  1000*hitsum/duration << endl;

   //Mode 3 Doesnt work
   // checkCUDAError(hipMalloc(&drH, w*h*sizeof(redHits[0])));
  
   // //copy from host to cuda memory
   
   // checkCUDAError(hipMemcpy(drH, &redHits[0], w*h*sizeof(redHits[0]), hipMemcpyHostToDevice));
  
   // generate_hits_prototype_kernel<<<1,256*256>>>(drH, w, h);
   
   // checkLastCUDAError_noAbort("kernelA");		   
   
   // checkCUDAError(hipMemcpy(&redHits[0], drH, w*h*sizeof(redHits[0]), hipMemcpyDeviceToHost));
   
   // checkCUDAError(hipFree(drH));
  
   // end = chrono::high_resolution_clock::now();
   // //cout << "sample time " << tix << " " << chrono::duration_cast<chrono::milliseconds>(end - start).count() << " ms" << endl; 
   
   // hitsum = 0;
   // for (auto e: redHits) {
   //   hitsum += e; }

   // duration = chrono::duration_cast<chrono::milliseconds>(end - start).count();
   // cout << "cuda generated red hits for <<<1,256*256>>>: " << hitsum << " in " << duration <<
   //     " ms. hits per second: " <<  1000*hitsum/duration << endl;
   
   return 0;
   
}


//Helper functions for buddhabrot
__device__ void generate_buddhabrot_trail_cuda(const hipDoubleComplex &c, unsigned int iters_max,
                                               hipDoubleComplex * p_trail, unsigned int &trail_len, unsigned long long & in, unsigned long long & out) {
  unsigned int iter_ix = 0;
  hipDoubleComplex z = make_hipDoubleComplex(0.0, 0.0);

  trail_len=0;

  while (iter_ix < iters_max && hipCabs(z) < 2.0) {
    z = hipCadd(hipCmul(z,z), c);

    p_trail[iter_ix] = z;
    ++iter_ix;
  }

  // If point is in the set we wont use it to color
  if (iter_ix == iters_max) {
    ++in;
    trail_len=0;
  }
  else
  {
    ++out;
    trail_len = iter_ix;
  }
  // return trail
}

__device__ bool skipInSet_cuda(hipDoubleComplex sample) {
  // if ((abs(sample - complex<double>(-1, 0)) < 0.25) ||
  //     (abs(1.0 - sqrt(1.0 - 4.0 * sample))) < 1.0)
  //Need equivalent math in cuda TODO missing sqrt
  if (hipCabs(hipCsub(sample,make_hipDoubleComplex(-1, 0))) < 0.25)
      return true;
    return false;
}

__device__ void saveBuddhabrotTrailToColor_cuda(hipDoubleComplex * p_trail, const unsigned int &trail_len, int w, int h,
                                                unsigned long long * p_hits, double minx, double maxx, double miny, double maxy) {
  int max_ix = w*h;

  for (int i = 0; i < trail_len; ++i) {
    // if point is plottable, scale it to be on a pixel and increment the
    // value for the pixel
    hipDoubleComplex c = p_trail[i];
    if ((hipCreal(c) <= maxx) && (hipCreal(c) >= minx) && (hipCimag(c) <= maxy) &&
        (hipCimag(c) >= miny)) {
      //depending on the cast here you might get a faint gridline in your image
      //so be careful
      int x = ((hipCreal(c) - minx) * w) /
          (maxx - minx);
      int y = ((hipCimag(c) - miny) * h) /
          (maxy - miny);
      
      int ix = x+y*w;
      
      //check for overrun
      if (ix < max_ix)
        atomicAdd(&p_hits[ix], 1);
    }
  }
}

//The actual kernel our fractals program uses
//2D array on device is one block of memory
__global__ void generate_hits_kernel(unsigned long long *rH, unsigned long long *gH, unsigned long long *bH, unsigned long long * p_stats,
                                     int w, int h, 
                                     double minx, double maxx, double miny, double maxy,
                                     int red_max, int green_max, int blue_max) {
  int i = threadIdx.x + blockDim.x * blockIdx.x; //dim is 1 and tix is 1
  
  if (i >= h*w)
    return;

  cuda_kernel_stats local_stats={0,0,0,0};

  hiprandState state;
  hiprand_init((unsigned long long)clock() + i, 0, 0, &state);


  unsigned long long max_samples = 100; //large enough to overcome thread sleep time and cuda overhead
  for (int sample_ix = 0; sample_ix < max_samples; ++sample_ix)
  {
    double pr = hiprand_uniform_double (&state);
    double pi = hiprand_uniform_double (&state);

    local_stats.total++;

    //scale the random sample    
    pr = minx + pr*(maxx - minx);
    pi = miny + pi*(maxy - miny);

    hipDoubleComplex sample = make_hipDoubleComplex(pr, pi);

    if (true == skipInSet_cuda(sample))
    {
      local_stats.rejected++;
      continue;
    }

    //We need memory to hold the escape trail - could be a problem
#define MAX_ITERS_CUDA (10000)
    hipDoubleComplex trail[MAX_ITERS_CUDA];
    unsigned int trail_len = 0;

    if ((red_max > MAX_ITERS_CUDA) || (green_max > MAX_ITERS_CUDA) || (blue_max > MAX_ITERS_CUDA))
    {
      printf("CUDA: Max iteration count not supported: %d %d %d\n", red_max, green_max, blue_max);
      return;;
    }
    

    generate_buddhabrot_trail_cuda(sample, red_max,
                                   &trail[0], trail_len, local_stats.in_set, local_stats.escaped_set);
    saveBuddhabrotTrailToColor_cuda(&trail[0], trail_len, w, h,
                                    rH, minx, maxx, miny, maxy);
    if (trail_len != 0) {
      sample = make_hipDoubleComplex(hipCreal(sample),-hipCimag(sample));
      generate_buddhabrot_trail_cuda(sample, red_max,
                                     &trail[0], trail_len, local_stats.in_set, local_stats.escaped_set);
      saveBuddhabrotTrailToColor_cuda(&trail[0], trail_len, w, h,
                                      rH, minx, maxx, miny, maxy);

    }

    generate_buddhabrot_trail_cuda(sample, green_max,
                                   &trail[0], trail_len, local_stats.in_set, local_stats.escaped_set);
    saveBuddhabrotTrailToColor_cuda(&trail[0], trail_len, w, h,
                                    gH, minx, maxx, miny, maxy);
    if (trail_len != 0) {
      sample = make_hipDoubleComplex(hipCreal(sample),-hipCimag(sample));
      generate_buddhabrot_trail_cuda(sample, green_max,
                                     &trail[0], trail_len, local_stats.in_set, local_stats.escaped_set);
      saveBuddhabrotTrailToColor_cuda(&trail[0], trail_len, w, h,
                                      gH, minx, maxx, miny, maxy);

    }

    generate_buddhabrot_trail_cuda(sample, blue_max,
                                   &trail[0], trail_len, local_stats.in_set, local_stats.escaped_set);
    saveBuddhabrotTrailToColor_cuda(&trail[0], trail_len, w, h,
                                    bH, minx, maxx, miny, maxy);
    if (trail_len != 0) {
      sample = make_hipDoubleComplex(hipCreal(sample),-hipCimag(sample));
      generate_buddhabrot_trail_cuda(sample, blue_max,
                                     &trail[0], trail_len, local_stats.in_set, local_stats.escaped_set);
      saveBuddhabrotTrailToColor_cuda(&trail[0], trail_len, w, h,
                                      bH, minx, maxx, miny, maxy);

    }
  }

  // now update p_stats for all samples for all kernels
  atomicAdd(&p_stats[0], local_stats.rejected);
  atomicAdd(&p_stats[1], local_stats.in_set);
  atomicAdd(&p_stats[2], local_stats.escaped_set);
  atomicAdd(&p_stats[3], local_stats.total);
}


//The main app will have one thread that will:
//1) Check for a future asking for thread termination
//2) Spawn a cuda kernel that takes about a second to run
//3) move the resulting Hits to the model under mutex
//4) back to 1)
//input: xyrange of fractal   pixel w and h  color max iterations
//output: stats

//kernel: move zeroed hits from host to device, skipInSet, generate trail, save trail in hits,
//move hits from device to host

int cuda_generate_buddhabrot_hits(unsigned int w, unsigned int h, SupportedFractal &frac,
				  SampleStats & stats,
                                  vector<vector<long long unsigned int>> &redHits,
                                  vector<vector<long long unsigned int>> &greenHits,
                                  vector<vector<long long unsigned int>> &blueHits)
{
  //spawn a kernel that takes ~ a second to run

  //cout << "CUDA Main: generate buddhabrot hits of all 3 colors" << endl;
  auto start = chrono::high_resolution_clock::now();

  //Zero out passed in hits
  redHits.resize(0);
  greenHits.resize(0);
  blueHits.resize(0);


  //Create 1D arrays of hits for cuda (its not good with C++ 2D vector<vector<>> style)
  vector<unsigned long long> rH;
  rH.resize(w*h,0);
  vector<unsigned long long> gH;
  gH.resize(w*h,0);
  vector<unsigned long long> bH;
  bH.resize(w*h,0);

  cuda_kernel_stats cuda_stats;


  unsigned long long * drH;
  unsigned long long * dgH;
  unsigned long long * dbH;

  unsigned long long * dstats;

  checkCUDAError(hipMalloc(&drH, w*h*sizeof(unsigned long long)));
  checkCUDAError(hipMalloc(&dgH, w*h*sizeof(unsigned long long)));
  checkCUDAError(hipMalloc(&dbH, w*h*sizeof(unsigned long long)));
  checkCUDAError(hipMalloc(&dstats, 4*sizeof(unsigned long long)));
  
  //copy from host to cuda memory
  
  checkCUDAError(hipMemcpy(drH, &rH[0], w*h*sizeof(unsigned long long), hipMemcpyHostToDevice));
  checkCUDAError(hipMemcpy(dgH, &gH[0], w*h*sizeof(unsigned long long), hipMemcpyHostToDevice));
  checkCUDAError(hipMemcpy(dbH, &bH[0], w*h*sizeof(unsigned long long), hipMemcpyHostToDevice));

  //64,256 is pretty fast
  generate_hits_kernel<<<32,256>>>(drH, dgH, dbH, dstats,
				  w, h,
				  frac.xMinMax[0], frac.xMinMax[1], frac.yMinMax[0], frac.yMinMax[1],
				  frac.max_iters[0], frac.max_iters[1], frac.max_iters[2]);
  
  checkLastCUDAError_noAbort("kernel for buddhabrot");		   
  
  checkCUDAError(hipMemcpy(&rH[0], drH, w*h*sizeof(unsigned long long), hipMemcpyDeviceToHost));
  checkCUDAError(hipMemcpy(&gH[0], dgH, w*h*sizeof(unsigned long long), hipMemcpyDeviceToHost));
  checkCUDAError(hipMemcpy(&bH[0], dbH, w*h*sizeof(unsigned long long), hipMemcpyDeviceToHost));

  checkCUDAError(hipMemcpy(&cuda_stats, dstats, 4*sizeof(unsigned long long), hipMemcpyDeviceToHost));
     
  checkCUDAError(hipFree(drH));
  checkCUDAError(hipFree(dgH));
  checkCUDAError(hipFree(dbH));
  checkCUDAError(hipFree(dstats));

  stats.total = cuda_stats.total;
  stats.rejected = cuda_stats.rejected;
  stats.in_set = cuda_stats.in_set;
  stats.escaped_set = cuda_stats.escaped_set;

  //cout << "cuda total/rejected/in_set/escaped_set " << stats.total << "/" << stats.rejected << "/" << stats.in_set << "/" << stats.escaped_set << endl;
  
  auto end = chrono::high_resolution_clock::now();
  //cout << "sample time " << tix << " " << chrono::duration_cast<chrono::milliseconds>(end - start).count() << " ms" << endl; 

  unsigned long long  hitsumr = 0;
  unsigned long long  hitsumg = 0;
  unsigned long long  hitsumb = 0;
  for (auto e: rH) {
    hitsumr += e; }
  for (auto e: gH) {
    hitsumg += e; }
  for (auto e: bH) {
    hitsumb += e; }

  auto duration = chrono::duration_cast<chrono::milliseconds>(end - start).count();
  //cout << "cuda generated rgb hits for <<<4*256,128>>>: " << hitsumr << " " << hitsumg << " " << hitsumb << " in " << duration <<
  //  " ms.   hits per second: " <<  1000*(hitsumr+hitsumg+hitsumb)/duration << endl;




   //Copy the 3 cuda 1D arrays into the user provided 2D arrays
   // width and heigh may be switched
   // The need for this code probably shows we need to redo our arrays
   redHits.resize(w);
   for (auto &v : redHits) v.resize(h);
   for (int i = 0; i < w; ++i)
   {
     for (int j = 0; j < h; ++j) {
       redHits[i][j] = rH[i+j*w];
     }
   }
   
   greenHits.resize(w);
   for (auto &v : greenHits) v.resize(h);
   for (int i = 0; i < w; ++i)
   {
     for (int j = 0; j < h; ++j) {
       greenHits[i][j] = gH[i+j*w];
     }
   }
   
   blueHits.resize(w);
   for (auto &v : blueHits) v.resize(h);
      for (int i = 0; i < w; ++i)
   {
     for (int j = 0; j < h; ++j) {
       blueHits[i][j] = bH[i+j*w];
     }
   }

   auto realend = chrono::high_resolution_clock::now();
   duration = chrono::duration_cast<chrono::milliseconds>(realend - end).count();
   //cout << "1D -> 2D array copy: " << duration << " ms" << endl;

  return 0;
}
